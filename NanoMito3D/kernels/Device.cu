/****************************************************************************
 *
 * Copyright (C) 2022 Fabian Hauser
 *
 * Author: Fabian Hauser <fabian.hauser@fh-linz.at>
 * University of Applied Sciences Upper Austria - Linz - Austra
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 ****************************************************************************/

#include "../Device.h"

#include <stdexcept>

namespace GPU {

static bool IS_DEVICE_AVAILABLE = false;
static bool INITIALIZED = false;

}

void GPU::initGPU() {
	if (INITIALIZED)
		return;
	double hTmp = 1;
	double *dTmp;
	hipMalloc(&dTmp, sizeof(double));
	hipMemcpy(dTmp, &hTmp, sizeof(double), hipMemcpyHostToDevice);
	hipFree(dTmp);

	int nDevices = 0;
	hipGetDeviceCount(&nDevices);
	IS_DEVICE_AVAILABLE = nDevices > 0;
}

bool GPU::isGPUAvailable()
{
	initGPU();
	return IS_DEVICE_AVAILABLE;
}

void GPU::cudaCheckError()
{
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		throw std::runtime_error(std::string("Cuda error ") + hipGetErrorName(err) + ": " + hipGetErrorString(err));
}

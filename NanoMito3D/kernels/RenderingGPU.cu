#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * Copyright (C) 2022 Fabian Hauser
 *
 * Author: Fabian Hauser <fabian.hauser@fh-linz.at>
 * University of Applied Sciences Upper Austria - Linz - Austra
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 ****************************************************************************/

#include "Rendering.h"
#include "Device.h"

#include <thrust/fill.h>
#include <thrust/device_vector.h>

#include <algorithm>

#define BLOCK_SIZE 1024

// original from https://forums.developer.nvidia.com/t/atomicmin-on-char-is-there-a-way-to-compare-char-to-in-to-use-atomicmin/22246/2
__device__ uint8_t atomicMaxU8(uint8_t* address, uint8_t val)
{
	unsigned int *base_address = (unsigned int *)((size_t)address & ~3);
	unsigned int selectors[] = {0x3214, 0x3240, 0x3410, 0x4210};
	unsigned int sel = selectors[(size_t)address & 3];
	unsigned int old, assumed, max_, new_;

	old = *base_address;
	do {
		assumed = old;
		max_ = max(val, (uint8_t)__byte_perm(old, 0, ((size_t)address & 3)));
		new_ = __byte_perm(old, max_, sel);
		old = atomicCAS(base_address, assumed, new_);
	} while (assumed != old);

	return old;
}

__global__ void drawPSF_kernel(uint8_t *dVolume, const Localization *dLocs, uint32_t n, const int3 volumeDims, const float3 voxelSize, int windowSize)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n)
		return;

	const Localization &l = dLocs[i];

	const int ix = static_cast<int>(std::round((l.x / voxelSize.x)));
	const int iy = static_cast<int>(std::round((l.y / voxelSize.y)));
	const int iz = static_cast<int>(std::round((l.z / voxelSize.z)));

	const size_t strideZ = volumeDims.x * volumeDims.y;

	const int w = windowSize/2;
	for (int z = -w; z <= w; ++z) {
		for (int y = -w; y <= w; ++y) {
			for (int x = -w; x <= w; ++x) {
				if ((ix + x < 0) || (iy + y < 0) || (iz + z < 0) ||
					 (ix + x >= volumeDims.x) || (iy + y >= volumeDims.y) || (iz + z >= volumeDims.z))
					continue;
				const float tx = ((ix + x) * voxelSize.x - l.x) / l.PAx;
				const float ty = ((iy + y) * voxelSize.y - l.y) / l.PAy;
				const float tz = ((iz + z) * voxelSize.z - l.z) / l.PAz;
				const float e = min((255.f/windowSize)*expf(-0.5f * tx * tx -0.5f * ty * ty -0.5f * tz * tz), 255.f);
				const uint8_t val = static_cast<uint8_t>(e);
				const size_t addr = (ix + x) + volumeDims.x * (iy + y) + strideZ * (iz + z);
				// safely write max to volume
				atomicMaxU8(dVolume + addr, val);
			}
		}
	}
}

Volume Rendering::render_gpu(Localizations &locs, std::array<float,3> voxelSize, int windowSize)
{
	int3 dims;
	dims.x = static_cast<int>(std::ceilf(locs.width()  / voxelSize[0]));
	dims.y = static_cast<int>(std::ceilf(locs.height() / voxelSize[1]));
	dims.z = static_cast<int>(std::ceilf(locs.depth()  / voxelSize[2]));

	Volume hVolume({dims.x, dims.y, dims.z}, voxelSize, {0.f, 0.f, locs.minZ()});

	thrust::device_vector<uint8_t> dVolume(hVolume.voxels());
	thrust::fill_n(dVolume.begin(), dVolume.size(), 0);

	// copy all localizations
	const uint32_t n = static_cast<uint32_t>(locs.size());
	const size_t bytes = locs.size() * sizeof(Localization);

	Localization *dLocs = nullptr;
	hipMalloc(&dLocs, bytes);
	hipMemcpy(dLocs, locs.data(), bytes, hipMemcpyHostToDevice);

	GPU::cudaCheckError();
	hipDeviceSynchronize();

	const dim3 block(BLOCK_SIZE);
	const dim3 grid((n + block.x - 1)/block.x);
	drawPSF_kernel<<<grid,block>>>(thrust::raw_pointer_cast(dVolume.data()),
											 dLocs, n,
											 dims,
											 make_float3(voxelSize[0], voxelSize[1], voxelSize[2]),
											 windowSize
											 );

	GPU::cudaCheckError();
	hipDeviceSynchronize();

	hipFree(dLocs);

	hipMemcpy(hVolume.data(), thrust::raw_pointer_cast(dVolume.data()), hVolume.voxels(), hipMemcpyDeviceToHost);

	GPU::cudaCheckError();
	hipDeviceSynchronize();

	return hVolume;
}

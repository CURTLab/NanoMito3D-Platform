#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * Copyright (C) 2022 Fabian Hauser
 *
 * Author: Fabian Hauser <fabian.hauser@fh-linz.at>
 * University of Applied Sciences Upper Austria - Linz - Austra
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 ****************************************************************************/

#include "LocalThreshold.h"
#include "Device.h"

#include <stdexcept>
#include <string>
#include <assert.h>

#define BLOCK_SIZE 256
#define BLOCK_SIZE2 16
#define BLOCK_SIZE3 8

//using idx_t = int64_t;
using idx_t = int;

HOST_DEV uint8_t LocalThreshold::otsuThreshold(const uint16_t hist[256], int numPixels)
{
	int i;
	uint8_t threshold = 0;
	const float term = 1.f / numPixels;

	float total_mean = 0.f;
	for (i = 0; i < 256; ++i)
		total_mean += i * term * hist[i];

	float max_bcv = 0.f;
	float cnh = 0.f;
	float mean = 0.f;
	for (i = 0; i < 256; ++i) {
		const float norm = term * hist[i];
		cnh += norm;
		mean += i * norm;

		float p = max(1E-7f, cnh);

		float bcv = total_mean * cnh - mean;
		bcv *= bcv / (p * (1.f - p));

		if (max_bcv < bcv) {
			max_bcv = bcv;
			threshold = i;
		}
	}
	return threshold;
}

HOST_DEV uint8_t LocalThreshold::isoDataThreshold(const uint16_t hist[256], int numPixels)
{
	int i;

	float toth = 0.f, h = 0.f;
	float totl = 0.f, l = 0.f;
	for (i = 1; i < 256; ++i) {
		toth += static_cast<float>(hist[i]);
		h += i * static_cast<float>(hist[i]);
	}

	uint8_t threshold = 255;
	for (i = 1; i < 255; ++i) {
		totl += hist[i];
		l += static_cast<float>(hist[i]) * i;
		toth -= hist[i+1];
		h -= static_cast<float>(hist[i+1]) * (i+1);
		if (totl > 0 && toth > 0 && i == (uint8_t)(0.5 * (l/totl + h/toth))) {
			threshold = i;
		}
	}
	return threshold;
}

#define VOLUMEFILTER_MAXWEIGHTS VOLUMEFILTER_MAXSIZE*VOLUMEFILTER_MAXSIZE*VOLUMEFILTER_MAXSIZE
__constant__ int32_t c_filterOffsets[VOLUMEFILTER_MAXWEIGHTS];

__global__ void local_threshold_kernel2D(LocalThreshold::Method method, const uint8_t *d_input, uint8_t *d_output, int z, int width, int height, int depth, int64_t voxels, int windowSize)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	const int64_t fsize = static_cast<int64_t>(windowSize) * windowSize * windowSize;
	const int64_t idx = static_cast<int64_t>(z) * width * height + static_cast<int64_t>(y) * width + static_cast<int64_t>(x);

	if (x < 0 || x >= width || y < 0 || y >= height || z < 0 || z >= depth)
		return;

	uint16_t hist[256];
	for (int i = 0; i < 256; ++i) hist[i] = 0;

	int num_pixels = fsize;
	const idx_t *f = c_filterOffsets;
	for (idx_t i = 0; i < fsize; ++i, f++) {
		const idx_t idx2 = idx + *f;
		if (idx2 >= 0 && idx2 < voxels)
			hist[d_input[idx2]]++;
		else
			hist[0]++;
	}

	if (method == LocalThreshold::Otsu)
		d_output[idx] = (d_input[idx] >= LocalThreshold::otsuThreshold(hist, num_pixels) ? 255 : 0);
	else if (method == LocalThreshold::IsoData)
		d_output[idx] = (d_input[idx] >= LocalThreshold::isoDataThreshold(hist, num_pixels) ? 255 : 0);
	else
		d_output[idx] = 0;
}

__global__ void local_threshold_kernel1D(LocalThreshold::Method method, const uint8_t *d_input, uint8_t *d_output, idx_t voxels, idx_t nFilter)
{
	const int block = blockIdx.x * blockDim.x;
	const int idx = block + threadIdx.x;

	if (idx >= voxels)
		return;

	uint16_t hist[256];
	for (int i = 0; i < 256; ++i) hist[i] = 0;

	for (idx_t i = 0; i < nFilter; ++i) {
		// index for 3D window
		const idx_t idx2 = idx + c_filterOffsets[i];
		const idx_t histIdx = (idx2 >= 0 && idx2 < voxels) ? d_input[idx2] : 0;
		hist[histIdx]++;
	}

	if (hist[0] == nFilter)
		d_output[idx] = 0;
	else if (method == LocalThreshold::Otsu)
		d_output[idx] = (d_input[idx] >= LocalThreshold::otsuThreshold(hist, nFilter) ? 255 : 0);
	else if (method == LocalThreshold::IsoData)
		d_output[idx] = (d_input[idx] >= LocalThreshold::isoDataThreshold(hist, nFilter) ? 255 : 0);
	else
		d_output[idx] = 0;
}

void LocalThreshold::localThrehsold_gpu(Method method, const Volume &input, Volume &output, int windowSize)
{
	// check output dims
	if ((input.width() != output.width()) ||
		(input.height() != output.height()) ||
		(input.depth() != output.depth())) {
		// realloc output if dims are different
		output = Volume(input.size(), input.voxelSize(), input.origin());
	}
	uint8_t *d_output = nullptr;
	hipMalloc(&d_output, input.voxels());

	uint8_t *d_input = nullptr;
	hipMalloc(&d_input, input.voxels());
	hipMemcpy(d_input, input.constData(), input.voxels(), hipMemcpyHostToDevice);

	if (windowSize > VOLUMEFILTER_MAXSIZE)
		throw std::runtime_error("Max filter size is " + std::to_string(VOLUMEFILTER_MAXSIZE) + "!");

	idx_t nFilter = static_cast<idx_t>(windowSize) * windowSize * windowSize;
	idx_t *filterOffsets = new idx_t[nFilter];
	idx_t *idx = filterOffsets;
	const idx_t r = windowSize/2;
	for (idx_t k = -r; k <= r; ++k) {
		for (idx_t j = -r; j <= r; ++j) {
			for (idx_t i = -r; i <= r; ++i)
				*idx++ = i + j * input.width() + k * input.width() * input.height();
		}
	}
	hipMemcpyToSymbol(HIP_SYMBOL(c_filterOffsets), filterOffsets, windowSize * windowSize * windowSize * sizeof(idx_t));
	delete [] filterOffsets;

	idx_t voxels = static_cast<idx_t>(input.voxels());
#if 1
	// 1D kernel
	int batchSize = BLOCK_SIZE * 1024;
	const dim3 block(BLOCK_SIZE);
	const dim3 grid((static_cast<uint32_t>(batchSize) + block.x - 1)/block.x);
	for (idx_t i = 0; i < voxels; i += batchSize)
		local_threshold_kernel1D<<<grid,block>>>(method, d_input + i, d_output + i, voxels - i, nFilter);
#else
	const size_t zStride = static_cast<size_t>(input.width()) * input.height();
	// 2D kernel
	const dim3 block(BLOCK_SIZE2, BLOCK_SIZE2);
	const dim3 grid((input.width() + block.x - 1)/block.x,
					(input.height() + block.y - 1)/block.y);

	for (int i = 0; i < input.depth(); ++i)
		local_threshold_kernel2D<<<grid,block>>>(method, d_input, d_output, i, input.width(), input.height(), input.depth(), voxels, windowSize);
#endif

	hipMemcpy(output.data(), d_output, input.voxels(), hipMemcpyDeviceToHost);
	GPU::cudaCheckError();

	hipFree(c_filterOffsets);
	hipFree(d_output);
	hipFree(d_input);
}

#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * Copyright (C) 2022 Fabian Hauser
 *
 * Author: Fabian Hauser <fabian.hauser@fh-linz.at>
 * University of Applied Sciences Upper Austria - Linz - Austra
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 ****************************************************************************/

#include "GaussianFilter.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Device.h"

#define BLOCK_SIZE 16
#define BLOCK_SIZE3 8

#define M_PIF 3.141592653589793238462643383279502884e+00f

__global__ void generateGaussian_kernel(float *d_kernel, int size, float sigma)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;
	if (x >= size || y >= size || z >= size)
		return;
	float g = 0.5f/(sigma*sigma);
	float f = 1.f/(2.f*M_PIF*sigma*sigma);
	float i = x - size/2;
	float j = y - size/2;
	float k = z - size/2;
	d_kernel[x + y * size] = f * exp(-g*i*i-g*j*j-g*k*k);
}

__global__ void filter3D_kernel(const uint8_t *d_input, uint8_t *d_output, int width, int height, int depth, int size, const float *d_kernel)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;
	int r = size/2;
	size_t idx;

	const size_t stride[3] = {1ull, static_cast<size_t>(width), static_cast<size_t>(width) * height};

	if (x >= width || y >= height || z >= depth)
		return;

	const float *dk = d_kernel;
	float sum = 0.f;
	float val = 0.f;
	for (int k = -r; k <= r; ++k) {
		int zi = z + k;
		for (int j = -r; j <= r; ++j) {
			int yi = y + j;
			for (int i = -r; i <= r; ++i, ++dk) {
				int xi = x + i;
				if (xi >= 0 && xi < width && yi >= 0 && yi < height && zi >= 0 && zi < depth) {
					idx = stride[0] * xi + stride[1] * yi + stride[2] * zi;
					val += d_input[idx] * (*dk);
					sum += *dk;
				}
			}
		}
	}

	idx = stride[0] * x + stride[1] * y + stride[2] * z;
	d_output[idx] = val / sum;
}

void GaussianFilter::gaussianFilter_gpu(const uint8_t *input, uint8_t *output, int width, int height, int depth, int size, float sigma)
{
	const int bytes = width * height * depth;
	const int kbytes = size * size * size * sizeof(float);

	uint8_t *d_input, *d_output;
	float *d_kernel;
	hipMalloc(&d_input, bytes);
	hipMalloc(&d_output, bytes);
	hipMalloc(&d_kernel, kbytes);

	hipMemcpy(d_input, input, bytes, hipMemcpyHostToDevice);

	const dim3 block(BLOCK_SIZE3,BLOCK_SIZE3,BLOCK_SIZE3);

	const dim3 kgrid((size + block.x - 1)/block.x,
						  (size + block.y - 1)/block.y,
						  (size + block.z - 1)/block.z);
	generateGaussian_kernel<<<kgrid,block>>>(d_kernel, size, sigma);
	GPU::cudaCheckError();
	hipDeviceSynchronize();

	const dim3 grid((width + block.x - 1)/block.x,
						 (height + block.y - 1)/block.y,
						 (depth + block.z - 1)/block.z);
	filter3D_kernel<<<grid,block>>>(d_input, d_output, width, height, depth, size, d_kernel);
	GPU::cudaCheckError();
	hipDeviceSynchronize();

	hipMemcpy(output, d_output, bytes, hipMemcpyDeviceToHost);
	GPU::cudaCheckError();

	hipFree(d_input);
	hipFree(d_output);
	hipFree(d_kernel);
}

#include "../Device.h"

#include <stdexcept>

namespace GPU {

static bool IS_DEVICE_AVAILABLE = false;
static bool INITIALIZED = false;

}

void GPU::initGPU() {
	if (INITIALIZED)
		return;
	double hTmp = 1;
	double *dTmp;
	hipMalloc(&dTmp, sizeof(double));
	hipMemcpy(dTmp, &hTmp, sizeof(double), hipMemcpyHostToDevice);
	hipFree(dTmp);

	int nDevices = 0;
	hipGetDeviceCount(&nDevices);
	IS_DEVICE_AVAILABLE = nDevices > 0;
}

bool GPU::isGPUAvailable()
{
	initGPU();
	return IS_DEVICE_AVAILABLE;
}

void GPU::cudaCheckError()
{
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		throw std::runtime_error("CUDA error: " + std::string(hipGetErrorString(err)));
}

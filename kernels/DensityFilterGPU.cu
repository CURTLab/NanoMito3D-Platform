#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "DensityFilter.h"

#include <memory>
#include <algorithm>

#include "cuNSearch.h"

#if 0
__device__ float sqr(float x) { return x*x; }

__global__ void filter_kernel(const Localization *dLocs, size_t nLocs, size_t offset, size_t n, uint8_t *dFiltered, int minPts, float r2)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n)
		 return;

	const float x = dLocs[i + offset].x;
	const float y = dLocs[i + offset].y;
	const float z = dLocs[i + offset].z;

	size_t count = 0;
	for (size_t j = 0; j < nLocs; ++j) {
		const float dist = sqr(dLocs[j].x - x) + sqr(dLocs[j].y - y) + sqr(dLocs[j].z - z);
		count += size_t(dist < r2);
	}

	dFiltered[i + offset] = count < minPts;
}

// simple brute force density filter
Localizations::const_iterator DensityFilter::remove_gpu(Localizations &locs, int minPoints, float radius)
{
	std::unique_ptr<uint8_t[]> filtered(new uint8_t[locs.size()]);

	const size_t n = 32768; //65536;

	uint8_t *dFiltered = nullptr;
	hipMalloc(&dFiltered, locs.size());

	dim3 numThreads(1024);
	dim3 numBlocks((n + numThreads.x - 1) / numThreads.x);
	// minPoints + 1 to account for the point itself

	for (int i = 0; i < locs.size(); i += n) {
		const size_t m = std::min(n, locs.size() - i);
		filter_kernel<<<numBlocks, numThreads>>>(locs.constData(DeviceType::Device), locs.size(), i, m, dFiltered, minPoints + 1, radius * radius);
		GPU::cudaCheckError();
		hipDeviceSynchronize();
	}

	hipMemcpy(filtered.get(), dFiltered, locs.size(), hipMemcpyDeviceToHost);
	GPU::cudaCheckError();

	hipFree(dFiltered);


	return std::remove_if(locs.begin(), locs.end(), [&filtered,&locs](const Localization &l) -> bool {
		const size_t idx = static_cast<size_t>(&l - locs.data());
		return filtered[idx];
	});
}
#else

Localizations::const_iterator DensityFilter::remove_gpu(Localizations &locs, int minPoints, float radius)
{
	const size_t nLocs = locs.size();

	float *pts = new float[nLocs * 3];
	for (size_t i = 0; i < nLocs; ++i) {
		pts[3*i + 0] = locs[i].x;
		pts[3*i + 1] = locs[i].y;
		pts[3*i + 2] = locs[i].z;
	}

	cuNSearch::NeighborhoodSearch nsearch(radius);

	//Add point set from the test data
	auto pointSetIndex = nsearch.add_point_set(pts, nLocs, false, true);
	nsearch.find_neighbors();

	auto &pointSet = nsearch.point_set(pointSetIndex);

	const auto ret = std::remove_if(locs.begin(), locs.end(), [&locs,&pointSet,minPoints](const Localization &l) -> bool {
		const size_t idx = static_cast<size_t>(&l - locs.data());
		return pointSet.n_neighbors(0, idx) < minPoints;
	});

	return ret;
}
#endif
